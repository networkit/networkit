/*
 * GPUTools.cu
 *
 *  Created on: 21.03.2022
 *      Author: Fabian Brandt-Tumescheit, fabratu
 *              Lucas Petersen
 */

#include <networkit/auxiliary/GPUTools.hpp>
#include <networkit/auxiliary/Log.hpp>

namespace Aux {

namespace GPUTools {

bool initAndTestCUDA(int dev) {
    // Test result is only true if CUDA toolkit is found on the host and a device context could be established.
    bool result = false;

#ifdef __HIPCC__
    auto checkInitError = [](hipError_t error, std::string msg) {
        if (error != hipSuccess) {
            printf("%s: %d\n", msg.c_str(), error);
            return false;
        }
        return true;
    };

    hipDevice_t cuDevice;
    hipCtx_t cuContext;

    //initialize CUDA
    hipInit(0);
    
    if(checkInitError(hipDeviceGet(&cuDevice, dev), "cannot get device " + std::to_string(dev)) && checkInitError(hipCtxCreate(&cuContext, 0, cuDevice), "cannot create context")) {
        result = true;
    }
#endif
    (void)dev;
    return result;
}

void synchronizeCUDA() {
#ifdef __HIPCC__
    hipDeviceSynchronize();
#else
    WARN("NetworKit core was built without GPU-support. Therefore synchronizeCUDA() provides no functionality.");
#endif
}

void resetCUDA() {
#ifdef __HIPCC__
    hipDeviceReset();
#else
    WARN("NetworKit core was built without GPU-support. Therefore resetCUDA() provides no functionality.");
#endif
}

std::vector<std::string> getCUDADevices() {
    std::vector<std::string> devList(0);
    
#ifdef __HIPCC__  
    int devCount;
    hipGetDeviceCount(&devCount);
    hipDeviceProp_t props;
    for(int i = 0; i < devCount; i++) {
        hipGetDeviceProperties(&props, i);
        devList.push_back(props.name);
    }
#endif
    return devList;
}

#ifdef __HIPCC__ 
void checkCUDAError(hipError_t err) {

    if (hipSuccess != err) {
        throw std::runtime_error("CUDA Error = " + std::to_string(err) + ": " + hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
#endif

int getDeviceMaxCUDA() {
#ifdef __HIPCC__ 
	int devCount;
	checkCUDAError(hipGetDeviceCount(&devCount));
	hipDeviceProp_t prop;

	int maxcc=0, bestdev=0;
	for(int i=0; i<devCount; i++){
		checkCUDAError(hipGetDeviceProperties(&prop,i));
		if((prop.major + 0.1*prop.minor) > maxcc){
				maxcc = prop.major + 0.1*prop.minor;
				bestdev = i;
		}	
	}
	checkCUDAError(hipSetDevice(bestdev));
	checkCUDAError(hipGetDeviceProperties(&prop,bestdev));
	return bestdev;
#else
    WARN("NetworKit core was built without GPU-support. Therefore getDeviceMaxCUDA() always returns an invalid device index.");
    return -1;
#endif
}



} /* namespace GPUTools */

} // namespace Aux
